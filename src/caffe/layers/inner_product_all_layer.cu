#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/mpitask.hpp"

namespace caffe {

template <typename Dtype>
void InnerProductAllLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  hipDeviceSynchronize();
  MPI_Gather(bottom[0]->gpu_data(),bottom[0]->count(),MPI_FLOAT,bottom_temp_.mutable_gpu_data(),
      bottom[0]->count(),MPI_FLOAT,0,MPI_COMM_WORLD);
  hipDeviceSynchronize();
  if (Caffe::getThreadId() == 0) {
    const Dtype* bottom_data = bottom_temp_.gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
        bottom_data, weight, (Dtype)0., top_data);
    if (bias_term_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
          bias_multiplier_.gpu_data(),
          this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
    }
  }
}

template <typename Dtype>
void InnerProductAllLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (Caffe::getThreadId() == 0) {
    if (this->param_propagate_down_[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* bottom_data = bottom_temp_.gpu_data();
      // Gradient with respect to weight
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
          top_diff, bottom_data, (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
    if (bias_term_ && this->param_propagate_down_[1]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      // Gradient with respect to bias
      caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
          bias_multiplier_.gpu_data(), (Dtype)1.,
          this->blobs_[1]->mutable_gpu_diff());
    }
    if (propagate_down[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      // Gradient with respect to bottom data
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
          top_diff, this->blobs_[0]->gpu_data(), (Dtype)0.,
          bottom_temp_.mutable_gpu_diff());
    }
  }
  hipDeviceSynchronize();
  MpiTaskList<Dtype> *task_list = (MpiTaskList<Dtype> *)Caffe::getTaskList();
  task_list->wait_all_task();
  MPI_Scatter(bottom_temp_.gpu_diff(),bottom[0]->count(),MPI_FLOAT,bottom[0]->mutable_gpu_diff(),bottom[0]->count(),
      MPI_FLOAT,0,MPI_COMM_WORLD);
  hipDeviceSynchronize();
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductAllLayer);

}  // namespace caffe
